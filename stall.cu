#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BDIM 32

hipDevice_t   device;
hipCtx_t  context;
hipModule_t   module;
hipFunction_t function;

#define module_file "kernel.cubin"
#define kernel_name "arr_kernel"

void initCUDA()
{
    int deviceCount = 0;
    hipError_t err = hipInit(0);
    if (err == hipSuccess)
        hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    hipDeviceGet(&device, 0);
    char name[100];
    hipDeviceGetName(name, 100, device);
    printf("> Using device 0: %s\n", name);

    err = hipCtxCreate(&context, 0, device);
    if (err != hipSuccess) {
        fprintf(stderr, "* Error initializing the CUDA context.\n");
        hipCtxDestroy(context);
        exit(-1);
    }

    err = hipModuleLoad(&module, module_file);
    if (err != hipSuccess) {
        fprintf(stderr, "* Error loading the module %s\n", module_file);
        const char * str;
        hipDrvGetErrorString(err, &str);
        fprintf(stderr, "%s\n", str);
        hipCtxDestroy(context);
        exit(-1);
    }

    err = hipModuleGetFunction(&function, module, kernel_name);

    if (err != hipSuccess) {
        fprintf(stderr, "* Error getting kernel function %s\n", kernel_name);
        const char * str;
        hipDrvGetErrorString(err, &str);
        fprintf(stderr, "%s\n", str);
        hipCtxDestroy(context);
        exit(-1);
    }
}

int main() {
    int size = BDIM * 16 * sizeof(int);
    int *in = (int *)malloc(size);
    int *out = (int *)malloc(size);
    int *in_dev, *out_dev;
    initCUDA();
    hipMalloc(&in_dev, size);
    hipMalloc(&out_dev, size);
    for (int i = 0; i < BDIM; ++i)
        in[i] = i;
    hipMemcpy(in_dev, in, size, hipMemcpyHostToDevice);
    void * args[2] = {&in_dev, &out_dev};
    hipModuleLaunchKernel(function, 
            1, 1, 1,
            BDIM, 1, 1,
            0, 0, args, 0);
    // Test
    hipMemcpy(out, out_dev, size, hipMemcpyDeviceToHost);
    printf("%d\n",out[0]);
    return 0;
}
